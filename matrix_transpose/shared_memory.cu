#include "hip/hip_runtime.h"
//
// Created by chiendb on 11/08/24.
//

#include <iostream>
#include <random>
#include <ctime>


#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)

void check_cuda_error(hipError_t err, const char *const func, const char *const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "Cuda Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() check_last_cuda_error(__FILE__, __LINE__)

void check_last_cuda_error(const char *const file, const int line) {
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void transpose_cpu(int *source, int *target, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            target[j * M + i] = source[i * N + j];
        }
    }
}

template<class T>
double measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, unsigned int num_repeats = 100,
                          unsigned int num_warmups = 100) {
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (unsigned int i{0}; i < num_warmups; ++i) {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (unsigned int i{0}; i < num_repeats; ++i) {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    double const latency{(double) time / num_repeats};

    return latency;
}

double measure_memory_bandwidth(int M, int N, double latency) {
    return 2e-6 * M * N * 4 / latency;
}

__global__
void transpose_shared_memory(int *source, int *target, int M, int N) {
    __shared__ int tile[16][16];
    unsigned int ix, iy;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;
    tile[threadIdx.y][threadIdx.x] = source[iy * M + ix];
    __syncthreads();
    ix = blockDim.y * blockIdx.y + threadIdx.x;
    iy = blockDim.x * blockIdx.x + threadIdx.y;
    if (ix < M && iy < N) {
        target[iy * N + ix] = tile[threadIdx.x][threadIdx.y];
    }
}

__global__
void transpose_shared_memory_padding(int *source, int *target, int M, int N) {
    __shared__ int tile[16][17];
    unsigned int ix, iy;
    ix = blockDim.x * blockIdx.x + threadIdx.x;
    iy = blockDim.y * blockIdx.y + threadIdx.y;
    tile[threadIdx.y][threadIdx.x] = source[iy * M + ix];
    __syncthreads();
    ix = blockDim.y * blockIdx.y + threadIdx.x;
    iy = blockDim.x * blockIdx.x + threadIdx.y;
    if (ix < M && iy < N) {
        target[iy * N + ix] = tile[threadIdx.x][threadIdx.y];
    }
}

void launch_transpose_shared_memory(int *source, int *target, int M, int N, dim3 grid_dim, dim3 block_dim, hipStream_t stream) {
    transpose_shared_memory<<<grid_dim, block_dim, 0, stream>>>(source, target, M, N);
}

void launch_transpose_shared_memory_padding(int *source, int *target, int M, int N, dim3 grid_dim, dim3 block_dim, hipStream_t stream) {
    transpose_shared_memory_padding<<<grid_dim, block_dim, 0, stream>>>(source, target, M, N);
}

bool check_result(int *source, int *target, int M, int N) {
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            if (source[i * N + j] != target[j * M + i]) {
                return false;
            }
        }
    }
    return true;
}

int main(int argc, char **argv) {
    int kernel_type = 0;
    int M = 2048;
    int N = 2048;
    int bM = 16;
    int bN = 16;
    int *h_source, *h_target;
    int *d_source, *d_target;

    if (argc > 1) {
        kernel_type = std::stoi(argv[1]);
    }

    if (argc > 2) {
        bM = std::stoi(argv[2]);
    }

    if (argc > 3) {
        bN = std::stoi(argv[3]);
    }

    if (argc > 4) {
        M = std::stoi(argv[4]);
    }

    if (argc > 5) {
        N = std::stoi(argv[5]);
    }

    auto size = M * N;

    dim3 block_dim(bM, bN);
    dim3 grid_dim((M + bM - 1) / bM, (N + bN - 1) / bN);

    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    h_source = (int *) malloc(size * sizeof(int));
    h_target = (int *) malloc(size * sizeof(int));

    // Random number generator
    std::random_device rd;  // Obtain a random number from hardware
    std::mt19937 gen(rd()); // Seed the generator
    std::uniform_int_distribution<> distr(1, 1000000000); // Define the range

    // Generate and print a random integer
    for (int i = 0; i < size; ++i) {
        h_source[i] = distr(gen);
    }

    std::clock_t time_start = std::clock();
    transpose_cpu(h_source, h_target, M, N);
    std::clock_t time_end = std::clock();

    double latency_cpu = (double) (time_end - time_start) / CLOCKS_PER_SEC * 1000;
    auto memory_bandwidth_cpu = measure_memory_bandwidth(M, N, latency_cpu);
    std::cout << "Memory bandwidth for transpose on CPU : " << memory_bandwidth_cpu << std::endl;

    hipSetDevice(0);

    CHECK_CUDA_ERROR(hipMalloc((void **) &d_source, size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void **) &d_target, size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_source, h_source, size * sizeof(int), hipMemcpyHostToDevice, stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    double latency_gpu = 0;
    double memory_bandwidth = 0;
    int num_repeats = 100;
    int num_warmups = 10;

    if (kernel_type == 0 || kernel_type == 1) {
        CHECK_CUDA_ERROR(hipMemset(d_target, 0, size * sizeof(int)));
        std::function<void(hipStream_t)> bound_function_transpose_shared_memory{
            std::bind(launch_transpose_shared_memory, d_source, d_target, M, N, grid_dim, block_dim, stream)};
        latency_gpu = measure_performance(bound_function_transpose_shared_memory, stream, num_repeats, num_warmups);
        memory_bandwidth = measure_memory_bandwidth(M, N, latency_gpu);
        std::cout << "Memory bandwidth for transpose shared memory on GPU : " << memory_bandwidth << std::endl;

        CHECK_CUDA_ERROR(hipMemcpyAsync(h_target, d_target, size * sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        if (!check_result(h_source, h_target, M, N)) {
            std::cout << "Result is incorrect\n";
        }
    }

    if (kernel_type == 0 || kernel_type == 2) {
        CHECK_CUDA_ERROR(hipMemset(d_target, 0, size * sizeof(int)));
        std::function<void(hipStream_t)> bound_function_transpose_shared_memory_padding{
            std::bind(launch_transpose_shared_memory_padding, d_source, d_target, M, N, grid_dim, block_dim, stream)};
        latency_gpu = measure_performance(bound_function_transpose_shared_memory_padding, stream, num_repeats, num_warmups);
        memory_bandwidth = measure_memory_bandwidth(M, N, latency_gpu);
        std::cout << "Memory bandwidth for transpose shared memory padding  on GPU : " << memory_bandwidth << std::endl;

        CHECK_CUDA_ERROR(hipMemcpyAsync(h_target, d_target, size * sizeof(int), hipMemcpyDeviceToHost, stream));
        CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
        if (!check_result(h_source, h_target, M, N)) {
            std::cout << "Result is incorrect\n";
        }
    }

    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
    free(h_source);
    free(h_target);
    CHECK_CUDA_ERROR(hipFree(d_source));
    CHECK_CUDA_ERROR(hipFree(d_target));
    return 0;
}
