#include "hip/hip_runtime.h"
//
// Created by chiendb on 10/29/24.
//

#include <iostream>
#include <random>
#include <ctime>


#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)

void check_cuda_error(hipError_t err, const char *const func, const char *const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "Cuda Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() check_last_cuda_error(__FILE__, __LINE__)

void check_last_cuda_error(const char *const file, const int line) {
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

void reduce_cpu(int *a, int &sum, int size) {
    for (int i = 0; i < size; ++i) {
        sum += a[i];
    }
}

template<class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, unsigned int num_repeats = 100,
                          unsigned int num_warmups = 100) {
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (unsigned int i{0}; i < num_warmups; ++i) {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (unsigned int i{0}; i < num_repeats; ++i) {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / 1000 / num_repeats};

    return latency;
}

__global__
void reduce_interleaved_pair(int *a, int *s, int size) {
    auto gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid >= size) {
        return;
    }

    auto *idata = a + blockDim.x * blockIdx.x;

    for (auto offset = blockDim.x / 2; offset >= 1; offset >>= 1) {
        if (threadIdx.x < offset) {
            idata[threadIdx.x] += idata[threadIdx.x + offset];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        s[blockIdx.x] = idata[0];
    }
}

void launch_reduce_interleaved_pair(int *a, int *s, int size, int grid_dim, int block_dim, hipStream_t stream) {
    reduce_interleaved_pair<<<grid_dim, block_dim, 0, stream>>>(a, s, size);
}


bool check_result(int *h_s, const int target, int grid_dim) {
    int output = 0;
    for (int i = 0; i < grid_dim; ++i) {
        output += h_s[i];
    }
    if (output != target) {
        std::cout << "output: " << output << ", target: " << target << "\n";
    }
    return output == target;
}

int main(int argc, char **argv) {
    int size = 1 << 24;
    int *h_a, *h_s, sum = 0;
    int *d_a, *d_s;

    int block_dim = std::stoi(argv[1]);
    int grid_dim = (size + block_dim - 1) / block_dim;

    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    h_a = (int *) malloc(size * sizeof(int));
    h_s = (int *) malloc(grid_dim * sizeof(int));

    // Random number generator
    std::random_device rd;  // Obtain a random number from hardware
    std::mt19937 gen(rd()); // Seed the generator
    std::uniform_int_distribution<> distr(1, 10); // Define the range

    // Generate and print a random integer
    for (int i = 0; i < size; ++i) {
        h_a[i] = distr(gen);
    }

    std::clock_t time_start = std::clock();
    reduce_cpu(h_a, sum, size);
    std::clock_t time_end = std::clock();

    double latency_cpu = (double) (time_end - time_start) / CLOCKS_PER_SEC;
    std::cout << "Latency for sum array on CPU: " << latency_cpu << std::endl;

    hipSetDevice(0);

    CHECK_CUDA_ERROR(hipMalloc((void **) &d_a, size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void **) &d_s, grid_dim * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemcpyAsync(d_a, h_a, size * sizeof(int), hipMemcpyHostToDevice, stream));
    CHECK_CUDA_ERROR(hipMemset(d_s, 0, grid_dim * sizeof(int)));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    std::function<void(hipStream_t)> bound_function_reduce_interleaved_pair{
            std::bind(launch_reduce_interleaved_pair, d_a, d_s, size, grid_dim, block_dim, stream)};

    float const latency_gpu{measure_performance(bound_function_reduce_interleaved_pair, stream, 1, 0)};
    std::cout << "Latency for sum array neighbored on GPU, block_dim " << block_dim << ": " << latency_gpu << std::endl;
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_s, d_s, grid_dim * sizeof(int), hipMemcpyDeviceToHost, stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    bool success = check_result(h_s, sum, grid_dim);
    if (success) {
        std::cout << "Result is correct\n";
    } else {
        std::cout << "Result is incorrect\n";
    }

    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
    free(h_a);
    free(h_s);
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_s));
    return 0;
}
