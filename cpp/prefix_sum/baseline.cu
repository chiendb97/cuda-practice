#include "hip/hip_runtime.h"
//
// Created by chiendb on 3/4/24.
//

#include <iostream>
#include <random>
#include <ctime>
#include <gflags/gflags.h>

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_scan.cuh>
#include <cub/device/device_scan.cuh>

DEFINE_uint32(n, 1 << 20, "n");
DEFINE_uint32(block_dim, 256, "block dim");
DEFINE_uint32(num_warmups, 1, "num_warmups");
DEFINE_uint32(num_repeats, 1, "num_repeats");


#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)

void check_cuda_error(hipError_t err, const char *const func, const char *const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "Cuda Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() check_last_cuda_error(__FILE__, __LINE__)

void check_last_cuda_error(const char *const file, const int line) {
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template<class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, unsigned int num_repeats = 100,
                          unsigned int num_warmups = 100) {
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (unsigned int i{0}; i < num_warmups; ++i) {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (unsigned int i{0}; i < num_repeats; ++i) {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

void prefix_sum_cpu(const float *X, float *Y, size_t n) {
    Y[0] = 0;
    for (int i = 1; i < n; ++i) {
        Y[i] = X[i-1] + Y[i - 1];
    }
}

template<int block_dim, int items_per_thread, hipcub::BlockScanAlgorithm algorithm>
__global__
void prefix_block_sum(const float *__restrict__ X, float *Y, size_t N) {
    typedef hipcub::BlockLoad<float, block_dim, items_per_thread, hipcub::BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;
    typedef hipcub::BlockStore<float, block_dim, items_per_thread, hipcub::BLOCK_STORE_WARP_TRANSPOSE> BlockStoreT;

    typedef hipcub::BlockScan<float, block_dim, algorithm> BlockScanT;

    __shared__ union {
        typename  BlockLoadT::TempStorage load;
        typename BlockStoreT::TempStorage store;
        typename BlockScanT::TempStorage scan;
    } temp_storage;

    float data[items_per_thread];

    BlockLoadT{temp_storage.load}.Load(X, data);

    __syncthreads();

    float aggregate;
    BlockScanT{temp_storage.scan}.ExclusiveScan(data, data, aggregate);

    __syncthreads();

    BlockStoreT{temp_storage.store}.Store(Y, data);

    if (threadIdx.x == 0) {
        Y[block_dim * items_per_thread] = aggregate;
    }
}

void launch_prefix_sum(const float *d_X, float *d_output, int n,
                       int grid_dim, int block_dim, hipStream_t stream) {
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_X, d_output, n);
    hipMalloc(&d_temp_storage, temp_storage_bytes * sizeof(float));
    hipcub::DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_X, d_output, n);
}

bool check_result(float *output, float *target, size_t n, float eps = 1e-2) {
    for (int i = 0; i < n; ++i) {
        if (fabs(output[i] - target[i]) > eps) {
            return false;
        }
    }
    return true;
}

int main(int argc, char *argv[]) {
    gflags::ParseCommandLineFlags(&argc, &argv, true);

    size_t n = FLAGS_n;
    size_t block_dim = FLAGS_block_dim;
    size_t num_warmups = FLAGS_num_warmups;
    size_t num_repeats = FLAGS_num_repeats;

    float *h_X, *h_output, *h_target;
    float *d_X, *d_output;

    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    h_X = (float *) malloc(n * sizeof(float));
    h_output = (float *) malloc(n * sizeof(float));
    h_target = (float *) malloc(n * sizeof(float));

    // Random number generator
    std::random_device rd; // Obtain a random number from hardware
    std::mt19937 gen(rd()); // Seed the generator
    std::uniform_real_distribution<> distr(-1.f, 1.f); // Define the range

    // Generate and print a random integer
    for (int i = 0; i < n; ++i) {
        h_X[i] = distr(gen);
    }

    std::clock_t time_start = std::clock();
    prefix_sum_cpu(h_X, h_target, n);
    std::clock_t time_end = std::clock();

    double latency_cpu = (double) (time_end - time_start) / CLOCKS_PER_SEC * 1000;
    std::cout << "Latency for prefix sum on CPU: " << latency_cpu << std::endl;

    hipSetDevice(0);

    CHECK_CUDA_ERROR(hipMalloc((void **) &d_X, n * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **) &d_output, n * sizeof(float)));

    CHECK_CUDA_ERROR(hipMemcpyAsync(d_X, h_X, n * sizeof(float), hipMemcpyHostToDevice, stream));

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    size_t grid_dim = (n + block_dim - 1) / block_dim;
    std::function<void(hipStream_t)> bound_function_prefix_sum{
        std::bind(launch_prefix_sum, d_X, d_output, n, grid_dim, block_dim, stream)
    };

    float const latency_gpu{measure_performance(bound_function_prefix_sum, stream, num_repeats, num_warmups)};
    std::cout << "Latency for prefix sum on GPU, block_dim " << block_dim << ": " << latency_gpu << std::endl;
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_output, n * sizeof(float), hipMemcpyDeviceToHost, stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    if (check_result(h_output, h_target, n)) {
        std::cout << "Result is correct" << std::endl;
    } else {
        std::cout << "Result is incorrect" << std::endl;
    }

    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
    free(h_X);
    free(h_output);
    free(h_target);
    CHECK_CUDA_ERROR(hipFree(d_X));
    CHECK_CUDA_ERROR(hipFree(d_output));
    return 0;
}
