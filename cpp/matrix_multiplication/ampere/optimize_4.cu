#include "hip/hip_runtime.h"
//
// Created by root on 6/25/25.
//

#include <functional>
#include <iostream>
#include <random>
#include <ctime>
#include <hip/hip_fp16.h>
#include <gflags/gflags.h>

#include "host_utils.cuh"
#include "device_utils.cuh"

DEFINE_uint32(M, 512, "M");
DEFINE_uint32(N, 512, "N");
DEFINE_uint32(K, 512, "K");
DEFINE_double(alpha, 1.0, "alpha");
DEFINE_double(beta, 1.0, "beta");

DEFINE_bool(test, false, "test");
DEFINE_uint32(num_warmups, 0, "num_warmups");
DEFINE_uint32(num_repeats, 1, "num_repeats");


template<uint32_t BM, uint32_t BN, uint32_t BK, uint32_t WM, uint32_t WN, uint32_t WK, uint32_t NUM_THREADS>
__global__ void
matrix_multiplication(const uint32_t M, const uint32_t N, const uint32_t K,
                      const float alpha, half *A, half *B, const float beta, half *C, half *D) {
    constexpr uint32_t MMA_M = 16;
    constexpr uint32_t MMA_N = 8;
    constexpr uint32_t MMA_K = 8;

    const uint32_t A_stride = K;
    const uint32_t B_stride = N;
    const uint32_t CD_stride = N;

    constexpr unsigned int SWIZZLE_BITS_A = int_log2(BK / 8);
    constexpr unsigned int SWIZZLE_BITS_B = int_log2(BN / 8);
    constexpr unsigned int SWIZZLE_MASK_A = 0b1110000 << SWIZZLE_BITS_A;
    constexpr unsigned int SWIZZLE_MASK_B = 0b1110000 << SWIZZLE_BITS_B;

    const uint32_t mma_tiles_per_warp_m = WM / MMA_M;
    const uint32_t mma_tiles_per_warp_n = WN / MMA_N;
    const uint32_t mma_tiles_per_warp_k = WK / MMA_K;

    const uint32_t warp_tiles_per_block_k = BK / WK;

    const uint32_t num_block_tiles_k = K / BK;

    const uint32_t block_m = blockIdx.y;
    const uint32_t block_n = blockIdx.x;

    const uint32_t warp_m = threadIdx.y;
    const uint32_t warp_n = threadIdx.x / 32;

    // Todo: Cast shared memory to SharedStorage
    extern __shared__ half shared_memory[];
    half *A_block_smem = shared_memory;
    half *B_block_smem = &shared_memory[BM * BK];

    uint32_t acc_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][2];
    uint32_t A_register[mma_tiles_per_warp_m][mma_tiles_per_warp_k][2];
    uint32_t B_register[mma_tiles_per_warp_k][mma_tiles_per_warp_n];

    half (&acc_register_)[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4] =
            reinterpret_cast<half(&)[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4]>(acc_register);

    for (uint32_t mma_m = 0; mma_m < mma_tiles_per_warp_m; ++mma_m) {
        for (uint32_t mma_n = 0; mma_n < mma_tiles_per_warp_n; ++mma_n) {
            acc_register_[mma_m][mma_n][0] = 0;
            acc_register_[mma_m][mma_n][1] = 0;
            acc_register_[mma_m][mma_n][2] = 0;
            acc_register_[mma_m][mma_n][3] = 0;
        }
    }

    //////////////
    // mainloop //
    //////////////

    constexpr uint32_t A_reg_size = BM * BK / NUM_THREADS / 8;
    constexpr uint32_t B_reg_size = BK * BN / NUM_THREADS / 8;
    float4 A_gmem_cache_reg[A_reg_size];
    float4 B_gmem_cache_reg[B_reg_size];

    // Prefetch the first block tile of A, B into shared memory
    half *A_block_gmem = A + block_m * BM * A_stride;
    half *B_block_gmem = B + block_n * BN;

    tiled_mem_cpy_swizzle<BM, BK, NUM_THREADS, 8, SWIZZLE_BITS_A>(reinterpret_cast<float4 *>(A_block_gmem), reinterpret_cast<float4 *>(A_block_smem), A_stride);
    tiled_mem_cpy_swizzle<BK, BN, NUM_THREADS, 8, SWIZZLE_BITS_B>(reinterpret_cast<float4 *>(B_block_gmem), reinterpret_cast<float4 *>(B_block_smem), B_stride);

    for (uint32_t block_k = 1; block_k <= num_block_tiles_k; ++block_k) {
        __syncthreads();

        if (block_k != num_block_tiles_k) {
            A_block_gmem = A + block_m * BM * A_stride + block_k * BK;
            B_block_gmem = B + block_k * BK * B_stride + block_n * BN;

            tiled_mem_cpy_load<BM, BK, NUM_THREADS, 8, A_reg_size>(reinterpret_cast<float4 *>(A_block_gmem), A_gmem_cache_reg, A_stride);
            tiled_mem_cpy_load<BK, BN, NUM_THREADS, 8, B_reg_size>(reinterpret_cast<float4 *>(B_block_gmem), B_gmem_cache_reg, B_stride);
        }

        for (uint32_t warp_k = 0; warp_k < warp_tiles_per_block_k; ++warp_k) {
            half *A_warp_smem = A_block_smem + warp_m * WM * BK + warp_k * WK;
            half *B_warp_smem = B_block_smem + warp_k * WK * BN + warp_n * WN;

            auto A_warp_tile_byte_offset = cvta_to_shared_u32(A_warp_smem);
            auto B_warp_tile_byte_offset = cvta_to_shared_u32(B_warp_smem);

            // Load tiles of A to register
            for (uint32_t mma_m = 0; mma_m < mma_tiles_per_warp_m; ++mma_m) {
                for (uint32_t mma_k = 0; mma_k < mma_tiles_per_warp_k; ++mma_k) {
                    const uint32_t mma_tile_byte_offset = (mma_m * MMA_M * BK + mma_k * MMA_K) * sizeof(half);
                    const uint32_t thread_byte_offset = (threadIdx.x % MMA_M) * BK * sizeof(half);

                    uint32_t thread_offset_bytes =
                            A_warp_tile_byte_offset + mma_tile_byte_offset + thread_byte_offset;

                    thread_offset_bytes = thread_offset_bytes ^ ((thread_offset_bytes & SWIZZLE_MASK_A) >> SWIZZLE_BITS_A);

                    asm volatile (
                        "ldmatrix.sync.aligned.m8n8.x2.shared.b16 "
                        "{%0, %1}, [%2];"
                        : "=r"(A_register[mma_m][mma_k][0]), "=r"(A_register[mma_m][mma_k][1])
                        : "r"(thread_offset_bytes)
                    );
                }
            }

            // Load tiles of B to register
            for (uint32_t mma_k = 0; mma_k < mma_tiles_per_warp_k; ++mma_k) {
                for (uint32_t mma_n = 0; mma_n < mma_tiles_per_warp_n; ++mma_n) {
                    const uint32_t mma_tile_byte_offset = (mma_k * MMA_K * BN + mma_n * MMA_N) * sizeof(half);
                    const uint32_t thread_byte_offset = (threadIdx.x % MMA_K) * BN * sizeof(half);

                    uint32_t thread_offset_bytes =
                            B_warp_tile_byte_offset + mma_tile_byte_offset + thread_byte_offset;

                    thread_offset_bytes = thread_offset_bytes ^ ((thread_offset_bytes & SWIZZLE_MASK_B) >> SWIZZLE_BITS_B);

                    asm volatile (
                        "ldmatrix.sync.aligned.m8n8.x1.trans.shared.b16 "
                        "{%0}, [%1];"
                        : "=r"(B_register[mma_k][mma_n])
                        : "r"(thread_offset_bytes)
                    );
                }
            }

            // Outer product between mma tiles

            for (uint32_t mma_k = 0; mma_k < mma_tiles_per_warp_k; ++mma_k) {
                for (uint32_t mma_n = 0; mma_n < mma_tiles_per_warp_n; ++mma_n) {
                    for (uint32_t mma_m = 0; mma_m < mma_tiles_per_warp_m; ++mma_m) {
                        asm volatile (
                            "mma.sync.aligned.m16n8k8.row.col.f16.f16.f16.f16 "
                            "{%0, %1}, "
                            "{%2, %3}, "
                            "{%4}, "
                            "{%5, %6};"
                            : "=r"(acc_register[mma_m][mma_n][0]), "=r"(acc_register[mma_m][mma_n][1])
                            : "r"(A_register[mma_m][mma_k][0]), "r"(A_register[mma_m][mma_k][1]),
                            "r"(B_register[mma_k][mma_n]),
                            "r"(acc_register[mma_m][mma_n][0]), "r"(acc_register[mma_m][mma_n][1])
                        );
                    }
                }
            }
        }

        __syncthreads();

        if (block_k != num_block_tiles_k) {
            tiled_mem_cpy_swizzle_store<BM, BK, NUM_THREADS, 8, A_reg_size, SWIZZLE_BITS_A>(A_gmem_cache_reg, reinterpret_cast<float4 *>(A_block_smem));
            tiled_mem_cpy_swizzle_store<BK, BN, NUM_THREADS, 8, B_reg_size, SWIZZLE_BITS_B>(B_gmem_cache_reg, reinterpret_cast<float4 *>(B_block_smem));
        }
    }

    //////////////
    // epilogue //
    //////////////

    half alpha_ = (half) alpha;
    half beta_ = (half) beta;

    half C_register[mma_tiles_per_warp_m][mma_tiles_per_warp_n][4];

    half *C_block_gmem = C + block_m * BM * CD_stride + block_n * BN;
    half *C_warp_gmem = C_block_gmem + warp_m * WM * CD_stride + warp_n * WN;

    half *D_block_gmem = D + block_m * BM * CD_stride + block_n * BN;
    half *D_warp_gmem = D_block_gmem + warp_m * WM * CD_stride + warp_n * WN;

    for (uint32_t mma_m = 0; mma_m < mma_tiles_per_warp_m; ++mma_m) {
        for (uint32_t mma_n = 0; mma_n < mma_tiles_per_warp_n; ++mma_n) {
            half *C_mma_gmem = C_warp_gmem + mma_m * MMA_M * CD_stride + mma_n * MMA_N;
            ldmatrix_m16n8_gmem(C_mma_gmem, C_register[mma_m][mma_n], CD_stride * sizeof(half));

            acc_register_[mma_m][mma_n][0] = alpha_ * acc_register_[mma_m][mma_n][0] + beta_ * C_register[mma_m][mma_n][
                                                 0];
            acc_register_[mma_m][mma_n][1] = alpha_ * acc_register_[mma_m][mma_n][1] + beta_ * C_register[mma_m][mma_n][
                                                 1];
            acc_register_[mma_m][mma_n][2] = alpha_ * acc_register_[mma_m][mma_n][2] + beta_ * C_register[mma_m][mma_n][
                                                 2];
            acc_register_[mma_m][mma_n][3] = alpha_ * acc_register_[mma_m][mma_n][3] + beta_ * C_register[mma_m][mma_n][
                                                 3];
        }
    }

    for (uint32_t mma_m = 0; mma_m < mma_tiles_per_warp_m; ++mma_m) {
        for (uint32_t mma_n = 0; mma_n < mma_tiles_per_warp_n; ++mma_n) {
            half *D_mma_gmem = D_warp_gmem + mma_m * MMA_M * CD_stride + mma_n * MMA_N;
            stmatrix_m16n8(D_mma_gmem, acc_register_[mma_m][mma_n], CD_stride * sizeof(half));
        }
    }
}

void launch_matrix_multiplication(const uint32_t M, const uint32_t N, const uint32_t K,
                                  const float alpha, half *A, half *B, const float beta, half *C, half *D,
                                  hipStream_t stream) {
    constexpr uint32_t BM = 256;
    constexpr uint32_t BN = 256;
    constexpr uint32_t BK = 32;

    constexpr uint32_t WARPS_PER_BLOCK_M = 2;
    constexpr uint32_t WARPS_PER_BLOCK_N = 4;
    constexpr uint32_t WARPS_PER_BLOCK_K = 4;

    constexpr uint32_t WM = BM / WARPS_PER_BLOCK_M;
    constexpr uint32_t WN = BN / WARPS_PER_BLOCK_N;
    constexpr uint32_t WK = BK / WARPS_PER_BLOCK_K;

    const uint32_t BLOCK_M = M / BM;
    const uint32_t BLOCK_N = N / BN;

    constexpr uint32_t WARP_SIZE = 32;

    constexpr uint32_t THREAD_M = WARPS_PER_BLOCK_M;
    constexpr uint32_t THREAD_N = WARPS_PER_BLOCK_N * WARP_SIZE;
    constexpr uint32_t NUM_THREADS = THREAD_M * THREAD_N;

    constexpr uint32_t smem_size = (BM * BK + BN * BK) * sizeof(half);

    dim3 grid_dim(BLOCK_N, BLOCK_M);
    dim3 block_dim(THREAD_N, THREAD_M);

    CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(matrix_multiplication<BM), BN, BK, WM, WN, WK, NUM_THREADS>,
        hipFuncAttributeMaxDynamicSharedMemorySize,(BM + BN) * BK * sizeof(half)));

    matrix_multiplication<BM, BN, BK, WM, WN, WK, NUM_THREADS>
            <<<grid_dim, block_dim, smem_size, stream>>>(M, N, K, alpha, A, B, beta, C, D);
}

int main(int argc, char *argv[]) {
    hipSetDevice(0);
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    uint32_t M = FLAGS_M;
    uint32_t N = FLAGS_N;
    uint32_t K = FLAGS_K;
    auto alpha = static_cast<float>(FLAGS_alpha);
    auto beta = static_cast<float>(FLAGS_beta);

    bool test = FLAGS_test;
    uint32_t num_warmups = FLAGS_num_warmups;
    uint32_t num_repeats = FLAGS_num_repeats;

    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    auto [host_params, device_params] = setup_params<half>(M, N, K, alpha, beta);

    std::function<void(hipStream_t)> bound_function_matrix_multiplication{
        std::bind(launch_matrix_multiplication, device_params.M, device_params.N, device_params.K,
                  device_params.alpha, device_params.A, device_params.B,
                  device_params.beta, device_params.C, device_params.D, stream)
    };

    float const latency_gpu{
        measure_performance(bound_function_matrix_multiplication, stream, num_repeats, num_warmups)
    };

    std::cout << "Latency for matrix multiplication on GPU: " << latency_gpu << std::endl;

    hipDeviceSynchronize();
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));

    if (test) {
        std::clock_t time_start = std::clock();
        matrix_multiplication_cpu(host_params.M, host_params.N, host_params.K,
                                  host_params.alpha, host_params.A, host_params.B,
                                  host_params.beta, host_params.C, host_params.D);
        std::clock_t time_end = std::clock();

        double latency_cpu = (double) (time_end - time_start) / CLOCKS_PER_SEC * 1000;
        std::cout << "Latency for matrix multiplication on CPU: " << latency_cpu << std::endl;

        if (check_result(device_params.D, host_params.D, M, N, 1e-4)) {
            std::cout << "Result is correct" << std::endl;
        } else {
            std::cout << "Result is incorrect" << std::endl;
        }
    }

    free_params(host_params, device_params);
    return 0;
}
