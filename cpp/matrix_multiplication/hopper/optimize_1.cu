#include "hip/hip_runtime.h"
//
// Created by root on 7/15/25.
//

#include <functional>
#include <iostream>
#include <random>
#include <ctime>
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <mma.h>
#include <cudaTypedefs.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <gflags/gflags.h>

#include "device_utils.cuh"
#include "host_utils.cuh"

DEFINE_uint32(M, 512, "M");
DEFINE_uint32(N, 512, "N");
DEFINE_uint32(K, 512, "K");
DEFINE_double(alpha, 1.0, "alpha");
DEFINE_double(beta, 0.0, "beta");

DEFINE_bool(test, false, "test");
DEFINE_uint32(num_warmups, 0, "num_warmups");
DEFINE_uint32(num_repeats, 1, "num_repeats");

template <uint32_t BM, uint32_t BN, uint32_t BK>
__global__ void
matrix_multiplication(const uint32_t M, const uint32_t N, const uint32_t K,
                      const float alpha, half *A, half *B, const float beta, half *C, half *D,
                      CUtensorMap *A_tensor_map, CUtensorMap *B_tensor_map)
{
    constexpr uint32_t WGMMA_M = 64;
    constexpr uint32_t WGMMA_N = 256;
    constexpr uint32_t WGMMA_K = 16;

    const uint32_t block_n = blockIdx.x;
    const uint32_t block_m = blockIdx.y;

    const uint32_t warp_group_idx = threadIdx.x / 128;

    __shared__ alignas(128) half A_block_smem[BM * BK];
    __shared__ alignas(128) half B_block_smem[BN * BK];

    constexpr uint32_t A_block_smem_num_bytes = BM * BK * sizeof(half);
    constexpr uint32_t B_block_smem_num_bytes = BN * BK * sizeof(half);

    __shared__ cuda::barrier<cuda::thread_scope_block> bar_A;
    __shared__ cuda::barrier<cuda::thread_scope_block> bar_B;

    if (threadIdx.x == 0)
    {
        init(&bar_A, blockDim.x);
        init(&bar_B, blockDim.x);

        cuda::device::experimental::fence_proxy_async_shared_cta();
    }

    __syncthreads();

    float accum_reg[128];
    memset(accum_reg, 0, sizeof(accum_reg));

    cuda::barrier<cuda::thread_scope_block>::arrival_token token_A, token_B;

    for (uint32_t block_k = 0; block_k < K / BK; ++block_k)
    {
        if (threadIdx.x == 0)
        {
            cuda::device::experimental::cp_async_bulk_tensor_2d_global_to_shared(&A_block_smem, A_tensor_map, block_k * BK, block_m * BM, bar_A);
            token_A = cuda::device::barrier_arrive_tx(bar_A, 1, A_block_smem_num_bytes);

            cuda::device::experimental::cp_async_bulk_tensor_2d_global_to_shared(&B_block_smem, B_tensor_map, block_k * BK, block_n * BN, bar_B);
            token_B = cuda::device::barrier_arrive_tx(bar_B, 1, B_block_smem_num_bytes);
        }
        else
        {
            token_A = bar_A.arrive();
            token_B = bar_B.arrive();
        }

        bar_A.wait(std::move(token_A));
        bar_B.wait(std::move(token_B));

        __syncthreads();

        warp_group_arrive();

        half *A_warp_group_smem = A_block_smem + warp_group_idx * WGMMA_M * BK;

#pragma unroll
        for (uint32_t i = 0; i < BK / WGMMA_K; ++i)
        {
            uint32_t offset = i * WGMMA_K;
            wgmma_m64n256k16_f32_f16_f16<BK * sizeof(half)>(accum_reg, A_warp_group_smem + offset, B_block_smem + offset);
        }

        wgmma_commit_group();
        wgmma_wait_group<0>();
    }

    half *C_block = C + (block_m * BM + warp_group_idx * WGMMA_M) * N + block_n * BN;

    uint32_t thread = threadIdx.x % 128;
    uint32_t warp = thread / 32;
    uint32_t lane = thread % 32;

    uint32_t row = (warp * 16) + (lane / 4);
    uint32_t col = (thread % 4) * 2;

#define OUT_IDX(i, j) (i) * N + (j)

    for (uint32_t column_group = 0; column_group < WGMMA_N / 16; ++column_group)
    {
        C_block[OUT_IDX(row, col)] = __float2half(accum_reg[column_group * 8]);
        C_block[OUT_IDX(row, col + 1)] = __float2half(accum_reg[column_group * 8 + 1]);
        C_block[OUT_IDX(row + 8, col)] = __float2half(accum_reg[column_group * 8 + 2]);
        C_block[OUT_IDX(row + 8, col + 1)] = __float2half(accum_reg[column_group * 8 + 3]);
        C_block[OUT_IDX(row, col + 8)] = __float2half(accum_reg[column_group * 8 + 4]);
        C_block[OUT_IDX(row, col + 9)] = __float2half(accum_reg[column_group * 8 + 5]);
        C_block[OUT_IDX(row + 8, col + 8)] = __float2half(accum_reg[column_group * 8 + 6]);
        C_block[OUT_IDX(row + 8, col + 9)] = __float2half(accum_reg[column_group * 8 + 7]);
        col += 16;
    }
#undef OUT_IDX
}

void launch_matrix_multiplication(const uint32_t M, const uint32_t N, const uint32_t K,
                                  const float alpha, half *A, half *B, const float beta, half *C, half *D,
                                  CUtensorMap *A_tensor_map_device, CUtensorMap *B_tensor_map_device,
                                  hipStream_t stream)
{
    constexpr uint32_t num_consumner_warp_groups = 1;
    constexpr uint32_t BM = 64 * num_consumner_warp_groups;
    constexpr uint32_t BN = 256;
    constexpr uint32_t BK = 64;

    create_tensor_map<BM, BK>(A, M, K, A_tensor_map_device);
    create_tensor_map<BN, BK>(B, N, K, B_tensor_map_device);

    const uint32_t BLOCK_M = (M + BM - 1) / BM;
    const uint32_t BLOCK_N = (N + BN - 1) / BN;

    constexpr uint32_t NUM_THREADS_PER_WARP_GROUP = 128;

    dim3 grid_dim(BLOCK_N, BLOCK_M);
    dim3 block_dim(NUM_THREADS_PER_WARP_GROUP * num_consumner_warp_groups);

    constexpr uint32_t smem_size = 2 * (BM * BK + BN * BK) * sizeof(half);

    CHECK_CUDA_ERROR(hipFuncSetAttribute(reinterpret_cast<const void*>(matrix_multiplication<BM), BN, BK>, hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));

    matrix_multiplication<BM, BN, BK><<<grid_dim, block_dim, smem_size, stream>>>(M, N, K, alpha, A, B, beta, C, D, A_tensor_map_device, B_tensor_map_device);
}

int main(int argc, char *argv[])
{
    hipSetDevice(0);
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    uint32_t M = FLAGS_M;
    uint32_t N = FLAGS_N;
    uint32_t K = FLAGS_K;
    auto alpha = static_cast<float>(FLAGS_alpha);
    auto beta = static_cast<float>(FLAGS_beta);

    bool test = FLAGS_test;
    uint32_t num_warmups = FLAGS_num_warmups;
    uint32_t num_repeats = FLAGS_num_repeats;

    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    auto [host_params, device_params] = setup_params<half>(M, N, K, alpha, beta);

    CUtensorMap *A_tensor_map_device = nullptr;
    CUtensorMap *B_tensor_map_device = nullptr;

    CHECK_CUDA_ERROR(hipMalloc(&A_tensor_map_device, sizeof(CUtensorMap)));
    CHECK_CUDA_ERROR(hipMalloc(&B_tensor_map_device, sizeof(CUtensorMap)));

    std::function<void(hipStream_t)> bound_function_matrix_multiplication{
        std::bind(launch_matrix_multiplication, device_params.M, device_params.N, device_params.K,
                  device_params.alpha, device_params.A, device_params.B,
                  device_params.beta, device_params.C, device_params.D,
                  A_tensor_map_device, B_tensor_map_device, stream)};

    float const latency_gpu{
        measure_performance(bound_function_matrix_multiplication, stream, num_repeats, num_warmups)};

    std::cout << "Latency for matrix multiplication on GPU: " << latency_gpu << std::endl;

    hipDeviceSynchronize();
    CHECK_CUDA_ERROR(hipStreamDestroy(stream));

    if (test)
    {
        std::clock_t time_start = std::clock();
        matrix_multiplication_cpu(host_params.M, host_params.N, host_params.K,
                                  host_params.alpha, host_params.A, host_params.B,
                                  host_params.beta, host_params.C, host_params.D);
        std::clock_t time_end = std::clock();

        double latency_cpu = (double)(time_end - time_start) / CLOCKS_PER_SEC * 1000;
        std::cout << "Latency for matrix multiplication on CPU: " << latency_cpu << std::endl;

        if (check_result(device_params.C, host_params.D, M, N, 1e-4))
        {
            std::cout << "Result is correct" << std::endl;
        }
        else
        {
            std::cout << "Result is incorrect" << std::endl;
        }
    }

    free_params(host_params, device_params);
    CHECK_CUDA_ERROR(hipFree(A_tensor_map_device));
    CHECK_CUDA_ERROR(hipFree(B_tensor_map_device));
    return 0;
}
