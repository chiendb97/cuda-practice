#include "hip/hip_runtime.h"
//
// Created by chiendb on 3/4/24.
//

#include <functional>
#include <iostream>
#include <random>
#include <ctime>
#include <gflags/gflags.h>

#include <cub/block/block_reduce.cuh>

DEFINE_uint32(b, 4, "b");
DEFINE_uint32(n, 8192, "n");
DEFINE_uint32(block_dim, 256, "block dim");
DEFINE_uint32(num_warmups, 1, "num_warmups");
DEFINE_uint32(num_repeats, 1, "num_repeats");


#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)

void check_cuda_error(hipError_t err, const char *const func, const char *const file, const int line) {
    if (err != hipSuccess) {
        std::cerr << "Cuda Runtime Error at: " << file << ":" << line << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() check_last_cuda_error(__FILE__, __LINE__)

void check_last_cuda_error(const char *const file, const int line) {
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess) {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

template<class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, unsigned int num_repeats = 100,
                          unsigned int num_warmups = 100) {
    hipEvent_t start, stop;
    float time;

    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    for (unsigned int i{0}; i < num_warmups; ++i) {
        bound_function(stream);
    }

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    CHECK_CUDA_ERROR(hipEventRecord(start, stream));
    for (unsigned int i{0}; i < num_repeats; ++i) {
        bound_function(stream);
    }
    CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));
    CHECK_LAST_CUDA_ERROR();
    CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    float const latency{time / num_repeats};

    return latency;
}

void layer_norm_cpu(const float *X, float *Y, size_t b, size_t n, float eps) {
    for (int i = 0; i < b; ++i) {
        float sum = 0;
        for (int j = 0; j < n; ++j) {
            sum += X[i * n + j] * X[i * n + j];
        }

        float inv_rms = 1.0f / sqrt(sum / n + eps);

        for (int j = 0; j < n; ++j) {
            Y[i * n + j] = X[i * n + j] * inv_rms;
        }
    }
}

__forceinline__ __device__ void multiply_accumulate(const float4 &a, const float4 &b, float &sum) {
    sum += a.x * b.x;
    sum += a.y * b.y;
    sum += a.z * b.z;
    sum += a.w * b.w;
}

template<int block_size>
__global__
void layer_norm(const float4 *__restrict__ X, float4 *Y, size_t B, size_t N, float invN, float eps) {
    auto ti = blockIdx.x;
    auto di = threadIdx.x;
    auto stride = block_size;

    if (ti >= B) {
        return;
    }

    X += ti * N;
    float sum = 0;
    float4 vec;

    for (auto i = di; i < N; i += stride) {
        vec = X[i];
        multiply_accumulate(vec, vec, sum);
    }

    using BlockReduce = hipcub::BlockReduce<float, block_size>;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    sum = BlockReduce{temp_storage}.Sum(sum);

    __shared__ float shared_inv_rms;

    if (threadIdx.x == 0) {
        shared_inv_rms = rsqrtf(sum * invN + eps);
    }

    __syncthreads();

    float inv_rms = shared_inv_rms;
    Y += ti * N;

    for (auto i = di; i < N; i += stride) {
        vec = X[i];
        vec.x *= inv_rms;
        vec.y *= inv_rms;
        vec.z *= inv_rms;
        vec.w *= inv_rms;
        Y[i] = vec;
    }
}

template
__global__
void layer_norm<128>(const float4 *__restrict__ X, float4 *Y, size_t B, size_t N, float invN, float eps);

template
__global__
void layer_norm<256>(const float4 *__restrict__ X, float4 *Y, size_t B, size_t N, float invN, float eps);

template
__global__
void layer_norm<512>(const float4 *__restrict__ X, float4 *Y, size_t B, size_t N, float invN, float eps);

template
__global__
void layer_norm<1024>(const float4 *__restrict__ X, float4 *Y, size_t B, size_t N, float invN, float eps);

void launch_layer_norm(const float *d_X, float *d_output, size_t b, size_t n, float eps,
                     int grid_dim,
                     int block_dim, hipStream_t stream) {
    if (block_dim == 128) {
        layer_norm<128><<<grid_dim, block_dim, 0, stream>>>(reinterpret_cast<const float4 *>(d_X),
                                                          reinterpret_cast<float4 *>(d_output), b, n / 4, 1.0f / n,
                                                          eps);
    } else if (block_dim == 256) {
        layer_norm<256><<<grid_dim, block_dim, 0, stream>>>(reinterpret_cast<const float4 *>(d_X),
                                                          reinterpret_cast<float4 *>(d_output), b, n / 4, 1.0f / n,
                                                          eps);
    } else if (block_dim == 512) {
        layer_norm<512><<<grid_dim, block_dim, 0, stream>>>(reinterpret_cast<const float4 *>(d_X),
                                                          reinterpret_cast<float4 *>(d_output), b, n / 4, 1.0f / n,
                                                          eps);
    } else if (block_dim == 1024) {
        layer_norm<1024><<<grid_dim, block_dim, 0, stream>>>(reinterpret_cast<const float4 *>(d_X),
                                                           reinterpret_cast<float4 *>(d_output), b, n / 4, 1.0f / n,
                                                           eps);
    } else {
        std::cerr << "Kernel does not support block dim: " << block_dim << std::endl;
    }
}

bool check_result(float *output, float *target, size_t n, float eps = 1e-2) {
    for (int i = 0; i < n; ++i) {
        if (fabs(output[i] - target[i]) > eps) {
            return false;
        }
    }
    return true;
}

int main(int argc, char *argv[]) {
    gflags::ParseCommandLineFlags(&argc, &argv, true);

    size_t b = FLAGS_b;
    size_t n = FLAGS_n;
    size_t block_dim = FLAGS_block_dim;
    size_t num_warmups = FLAGS_num_warmups;
    size_t num_repeats = FLAGS_num_repeats;
    const float eps = 1e-5;

    float *h_X, *h_output, *h_target;
    float *d_X, *d_output;

    hipStream_t stream;
    CHECK_CUDA_ERROR(hipStreamCreate(&stream));

    h_X = (float *) malloc(b * n * sizeof(float));
    h_output = (float *) malloc(b * n * sizeof(float));
    h_target = (float *) malloc(b * n * sizeof(float));

    // Random number generator
    std::random_device rd; // Obtain a random number from hardware
    std::mt19937 gen(rd()); // Seed the generator
    std::uniform_real_distribution<> distr(0.f, 1.f); // Define the range

    // Generate and print a random integer
    for (int i = 0; i < b * n; ++i) {
        h_X[i] = distr(gen);
    }

    std::clock_t time_start = std::clock();
    layer_norm_cpu(h_X, h_target, b, n, eps);
    std::clock_t time_end = std::clock();

    double latency_cpu = (double) (time_end - time_start) / CLOCKS_PER_SEC * 1000;
    std::cout << "Latency for layer norm on CPU: " << latency_cpu << std::endl;

    hipSetDevice(0);

    CHECK_CUDA_ERROR(hipMalloc((void **) &d_X, b * n * sizeof(float)));
    CHECK_CUDA_ERROR(hipMalloc((void **) &d_output, b * n * sizeof(float)));

    CHECK_CUDA_ERROR(hipMemcpyAsync(d_X, h_X, b * n * sizeof(float), hipMemcpyHostToDevice, stream));

    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

    size_t grid_dim = b;
    std::function<void(hipStream_t)> bound_function_layer_norm{
        std::bind(launch_layer_norm, d_X, d_output, b, n, eps, grid_dim, block_dim, stream)
    };

    float const latency_gpu{measure_performance(bound_function_layer_norm, stream, num_repeats, num_warmups)};
    std::cout << "Latency for layer norm on GPU, block_dim " << block_dim << ": " << latency_gpu << std::endl;
    CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_output, b * n * sizeof(float), hipMemcpyDeviceToHost, stream));
    CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
    bool success = check_result(h_output, h_target, b * n);
    if (success) {
        std::cout << "Result is correct" << std::endl;
    } else {
        std::cout << "Result is incorrect" << std::endl;
    }

    CHECK_CUDA_ERROR(hipStreamDestroy(stream));
    free(h_X);
    free(h_output);
    free(h_target);
    CHECK_CUDA_ERROR(hipFree(d_X));
    CHECK_CUDA_ERROR(hipFree(d_output));
    return 0;
}
