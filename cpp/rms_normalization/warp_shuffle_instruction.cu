#include "hip/hip_runtime.h"
//
// Created by chiendb on 3/4/24.
//

#include <ctime>
#include <functional>
#include <gflags/gflags.h>
#include <iostream>
#include <random>

#include <cub/block/block_reduce.cuh>

DEFINE_uint32(b, 4, "b");
DEFINE_uint32(n, 8192, "n");
DEFINE_uint32(block_dim, 256, "block dim");
DEFINE_uint32(num_warmups, 1, "num_warmups");
DEFINE_uint32(num_repeats, 1, "num_repeats");

#define CHECK_CUDA_ERROR(val) check_cuda_error((val), #val, __FILE__, __LINE__)

void check_cuda_error(hipError_t err, const char *const func,
                      const char *const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "Cuda Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

#define CHECK_LAST_CUDA_ERROR() check_last_cuda_error(__FILE__, __LINE__)

void check_last_cuda_error(const char *const file, const int line) {
  hipError_t const err{hipGetLastError()};
  if (err != hipSuccess) {
    std::cerr << "CUDA Runtime Error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << std::endl;
    std::exit(EXIT_FAILURE);
  }
}

template <class T>
float measure_performance(std::function<T(hipStream_t)> bound_function,
                          hipStream_t stream, unsigned int num_repeats = 100,
                          unsigned int num_warmups = 100) {
  hipEvent_t start, stop;
  float time;

  CHECK_CUDA_ERROR(hipEventCreate(&start));
  CHECK_CUDA_ERROR(hipEventCreate(&stop));

  for (unsigned int i{0}; i < num_warmups; ++i) {
    bound_function(stream);
  }

  CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

  CHECK_CUDA_ERROR(hipEventRecord(start, stream));
  for (unsigned int i{0}; i < num_repeats; ++i) {
    bound_function(stream);
  }
  CHECK_CUDA_ERROR(hipEventRecord(stop, stream));
  CHECK_CUDA_ERROR(hipEventSynchronize(stop));
  CHECK_LAST_CUDA_ERROR();
  CHECK_CUDA_ERROR(hipEventElapsedTime(&time, start, stop));
  CHECK_CUDA_ERROR(hipEventDestroy(start));
  CHECK_CUDA_ERROR(hipEventDestroy(stop));

  float const latency{time / num_repeats};

  return latency;
}

void rms_norm_cpu(const float *X, float *Y, size_t b, size_t n, float eps) {
  for (int i = 0; i < b; ++i) {
    float sum = 0;
    for (int j = 0; j < n; ++j) {
      sum += X[i * n + j] * X[i * n + j];
    }

    float inv_rms = 1.0f / sqrt(sum / n + eps);

    for (int j = 0; j < n; ++j) {
      Y[i * n + j] = X[i * n + j] * inv_rms;
    }
  }
}

__forceinline__ __device__ void
multiply_accumulate(const float4 &a, const float4 &b, float &sum) {
  sum += a.x * b.x;
  sum += a.y * b.y;
  sum += a.z * b.z;
  sum += a.w * b.w;
}

template <typename T, int NUM>
__forceinline__ __device__ T warp_reduce_sum(T *val) {
#pragma unroll
  for (int i = 0; i < NUM; ++i) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
      val[i] += __shfl_xor_sync(0xffffffff, val[i], mask, 32);
    }
  }
  return (T)(0.0f);
}

template <typename T, int NUM>
__forceinline__ __device__ T block_reduce_sum(T *val) {
  __shared__ T shared[NUM][33];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;
  warp_reduce_sum<T, NUM>(val);

  if (lane == 0) {
#pragma unroll
    for (int i = 0; i < NUM; ++i) {
      shared[i][wid] = val[i];
    }
  }

  __syncthreads();

  bool is_mask = threadIdx.x < (blockDim.x / 32.f);
#pragma unroll
  for (int i = 0; i < NUM; ++i) {
    val[i] = is_mask ? shared[i][lane] : (T)(0.0f);
  }
  warp_reduce_sum<T, NUM>(val);

  return (T)(0.0f);
}

__global__ void rms_norm(const float4 *__restrict__ X, float4 *Y, size_t B,
                         size_t N, float invN, float eps) {
  auto ti = blockIdx.x;
  auto di = threadIdx.x;

  if (ti >= B) {
    return;
  }

  X += ti * N;
  float sum[1] = {0};

  float4 vec;

  // #pragma unroll 4
  for (auto i = di; i < N; i += blockDim.x) {
    vec = __ldg(&X[i]);
    sum[0] += vec.x * vec.x;
    sum[0] += vec.y * vec.y;
    sum[0] += vec.z * vec.z;
    sum[0] += vec.w * vec.w;
  }

  if (blockDim.x < 32) {
    warp_reduce_sum<float, 1>(sum);
  } else {
    block_reduce_sum<float, 1>(sum);
  }

  __shared__ float shared_inv_rms;

  if (threadIdx.x == 0) {
    shared_inv_rms = rsqrtf(sum[0] * invN + eps);
  }

  __syncthreads();

  float inv_rms = shared_inv_rms;
  Y += ti * N;

  // #pragma unroll 4
  for (auto i = di; i < N; i += blockDim.x) {
    vec = __ldg(&X[i]);
    vec.x *= inv_rms;
    vec.y *= inv_rms;
    vec.z *= inv_rms;
    vec.w *= inv_rms;
    Y[i] = vec;
  }
}

void launch_rms_norm(const float *d_X, float *d_output, size_t b, size_t n,
                     float eps, int grid_dim, int block_dim,
                     hipStream_t stream) {
  rms_norm<<<grid_dim, block_dim, 0, stream>>>(
      reinterpret_cast<const float4 *>(d_X),
      reinterpret_cast<float4 *>(d_output), b, n / 4, 1.0f / n, eps);
}

bool check_result(float *output, float *target, size_t n, float eps = 1e-2) {
  for (int i = 0; i < n; ++i) {
    if (fabs(output[i] - target[i]) > eps) {
      return false;
    }
  }
  return true;
}

int main(int argc, char *argv[]) {
  gflags::ParseCommandLineFlags(&argc, &argv, true);

  size_t b = FLAGS_b;
  size_t n = FLAGS_n;
  size_t block_dim = FLAGS_block_dim;
  size_t num_warmups = FLAGS_num_warmups;
  size_t num_repeats = FLAGS_num_repeats;
  const float eps = 1e-5;

  float *h_X, *h_output, *h_target;
  float *d_X, *d_output;

  hipStream_t stream;
  CHECK_CUDA_ERROR(hipStreamCreate(&stream));

  h_X = (float *)malloc(b * n * sizeof(float));
  h_output = (float *)malloc(b * n * sizeof(float));
  h_target = (float *)malloc(b * n * sizeof(float));

  // Random number generator
  std::random_device rd;  // Obtain a random number from hardware
  std::mt19937 gen(rd()); // Seed the generator
  std::uniform_real_distribution<> distr(0.f, 1.f); // Define the range

  // Generate and print a random integer
  for (int i = 0; i < b * n; ++i) {
    h_X[i] = distr(gen);
  }

  std::clock_t time_start = std::clock();
  rms_norm_cpu(h_X, h_target, b, n, eps);
  std::clock_t time_end = std::clock();

  double latency_cpu = (double)(time_end - time_start) / CLOCKS_PER_SEC * 1000;
  std::cout << "Latency for rms norm on CPU: " << latency_cpu << std::endl;

  hipSetDevice(0);

  CHECK_CUDA_ERROR(hipMalloc((void **)&d_X, b * n * sizeof(float)));
  CHECK_CUDA_ERROR(hipMalloc((void **)&d_output, b * n * sizeof(float)));

  CHECK_CUDA_ERROR(hipMemcpyAsync(d_X, h_X, b * n * sizeof(float),
                                   hipMemcpyHostToDevice, stream));

  CHECK_CUDA_ERROR(hipStreamSynchronize(stream));

  size_t grid_dim = b;
  std::function<void(hipStream_t)> bound_function_rms_norm{std::bind(
      launch_rms_norm, d_X, d_output, b, n, eps, grid_dim, block_dim, stream)};

  float const latency_gpu{measure_performance(bound_function_rms_norm, stream,
                                              num_repeats, num_warmups)};
  std::cout << "Latency for rms norm on GPU, block_dim " << block_dim << ": "
            << latency_gpu << std::endl;
  CHECK_CUDA_ERROR(hipMemcpyAsync(h_output, d_output, b * n * sizeof(float),
                                   hipMemcpyDeviceToHost, stream));
  CHECK_CUDA_ERROR(hipStreamSynchronize(stream));
  bool success = check_result(h_output, h_target, b * n);
  if (success) {
    std::cout << "Result is correct" << std::endl;
  } else {
    std::cout << "Result is incorrect" << std::endl;
  }

  CHECK_CUDA_ERROR(hipStreamDestroy(stream));
  free(h_X);
  free(h_output);
  free(h_target);
  CHECK_CUDA_ERROR(hipFree(d_X));
  CHECK_CUDA_ERROR(hipFree(d_output));
  return 0;
}
